#include<stdio.h>
#include "hip/hip_runtime.h"

int main()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
  
    printf("device count is: %d\n", deviceCount);

    for (int dev = 0; dev < deviceCount; dev++)
    {
    	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

        printf("\ndevice name is %d %s:\n", dev, deviceProp.name);
        printf("total amount global memory is: %u bytes\n", deviceProp.totalGlobalMem);
        printf("total amount global memory is: %u M\n", deviceProp.totalGlobalMem / 1024);
        printf("total amount global memory is: %u G\n", deviceProp.totalGlobalMem / (1024*1024));
        printf("number of multiprocessor is: %d\n", deviceProp.multiProcessorCount);
        printf("max number of thread per block %d\n", deviceProp.maxThreadsPerBlock);
        printf("max size of each dimension %d, %d, %d \n",deviceProp.maxThreadsDim[0],
                    deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);

        printf("\n");	
    }
}
